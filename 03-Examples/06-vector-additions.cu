
#include <hip/hip_runtime.h>
#include <stdio.h>

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (idx < N)
    {
        result[idx] = a[idx] + b[idx];
    }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);
  size_t threads_per_block = 256;
  size_t number_of_blocks = (N+threads_per_block-1)/threads_per_block;

  float *a;
  float *b;
  float *c;

  // a = (float *)malloc(size);
  // b = (float *)malloc(size);
  // c = (float *)malloc(size);
  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);


  addVectorsInto<<<number_of_blocks, threads_per_block>>>(c,a,b,N);

  hipError_t syncErr = hipGetLastError();
  hipError_t asyncErr = hipDeviceSynchronize();
  

  if (syncErr != hipSuccess)
  {  
    // Error in the kernel execution
    printf("Sync Error: %s\n", hipGetErrorString(syncErr));
  }
  if (asyncErr != hipSuccess) 
  {
    printf("Async Error: %s\n", hipGetErrorString(asyncErr));
  }

  checkElementsAre(7, c, N);


  hipFree(a);
  hipFree(b);
  hipFree(c);
}